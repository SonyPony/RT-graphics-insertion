#include "hip/hip_runtime.h"
#include "vibe.cuh"
#include <iostream>

__global__ void addKernel(int *c, const int *a, const int *b)
{
	
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void gpu::add(int* a, int* b, int* dest, int arrSize) {
	int* d_a = nullptr;
	int* d_b = nullptr;
	int* d_result = nullptr;
	const size_t bytesArrSize = arrSize * sizeof(int);

	hipError_t cudaStatus = hipSetDevice(0);

	// alloc device mem
	hipMalloc(reinterpret_cast<void**>(&d_a), bytesArrSize);
	hipMalloc(reinterpret_cast<void**>(&d_b), bytesArrSize);
	hipMalloc(reinterpret_cast<void**>(&d_result), bytesArrSize);

	// copy values
	hipMemcpy(d_a, a, bytesArrSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, bytesArrSize, hipMemcpyHostToDevice);

	addKernel <<<1, arrSize>>> (d_result, d_a, d_b);
	hipDeviceSynchronize();
	hipMemcpy(dest, d_result, bytesArrSize, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
}

// ARGB
__global__ void segment(uint8_t* input, /*uint8_t* model, */uint8_t* dest) {
	constexpr int stride = 4;	// 4 bytes strides
	const int i = (blockDim.x * blockIdx.x + threadIdx.x) * stride;

	dest[i] = input[i];		// A
	dest[i + 1] = 0;//input[i + 1];		// R
	dest[i + 2] = input[i + 2];		// G
	dest[i + 3] = input[i + 3];		// B
	//printf("%d\n", i);
}

void GPU::InsertionGraphicsPipeline::process(uint8_t * input, uint8_t * graphics, uint8_t * dest) {
	using T = decltype(input);
	T d_input = nullptr;
	T d_dest = nullptr;
	const size_t dataSize = 4 * m_height * m_width;

	hipSetDevice(0);
	hipMalloc(reinterpret_cast<void**>(&d_input), dataSize);
	hipMalloc(reinterpret_cast<void**>(&d_dest), dataSize);

	hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice);
	dim3 blocks(m_width / 16, m_height / 16);
	dim3 threads(16, 16);

	segment <<<1280, 720 >>> (d_input, d_dest);

	std::cout << "ahoj" << std::endl;

	hipMemcpy(dest, d_dest, dataSize, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipFree(d_dest);
}