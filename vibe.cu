#include "hip/hip_runtime.h"
#include "vibe.cuh"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>

using RandState = uint32_t;

__global__ void segment_new(uint8_t* input, uint8_t* model, uint8_t* dest) {
	constexpr int stride = 4;	// 4 bytes strides
	const int i = (blockDim.x * blockIdx.x + threadIdx.x);
	const int pixel_i = i * stride;

	int count = 0;
	for (int j = 0; j < 20; j++) {
		const int sample_i = i * (20 * 3) + j * 3;
		float distance =
			norm3df(
				(float)input[pixel_i] - model[sample_i],
				(float)input[pixel_i + 1] - model[sample_i + 1],
				(float)input[pixel_i + 2] - model[sample_i + 2]);
		if (distance < 20)
			count++;
	}

	dest[i] = 0;
	if (count < 2)
		dest[i] = 255;

}

__device__ __forceinline__ float devRand(RandState & state) {
	constexpr RandState RNG_COEFF = 4164903690U;
	state = state * RNG_COEFF + (state >> 16);
	return static_cast<float>(state) / UINT32_MAX;
}

__global__ void segment_fast(uint8_t* input, uint8_t* model, uint8_t* dest, int size, int currentSample, RandState* randState) {
	constexpr int stride = 4;	// 4 bytes strides
	const int id = (blockDim.x * blockIdx.x + threadIdx.x);
	const int pixel_i = id * stride;

	int count = 0;
	for (int j = 0; j < 20; j++) {
		const int sample_i = size * j * 3;
		float distance = norm3df(
			(float)input[pixel_i] - model[id + sample_i],
			(float)input[pixel_i + 1] - model[id + sample_i + size],
			(float)input[pixel_i + 2] - model[id + sample_i + size * 2]
		);

		if (distance < 20)
			count++;
	}

	uint8_t result = 0;
	const int isBackground = (count >= 2);
	if (count < 2)
		result = 255;
	dest[id] = result;

	// update
	RandState localRandState = randState[id];
	float rand = devRand(localRandState) * 16;
	if (rand == 0) {
		for (int i = 0; i < 3; i++)
			model[id + currentSample + i * size] = input[pixel_i + i];
	}

	rand = devRand(localRandState) * 16;
	// todo change to something like stencil?
	if (rand == 0) {
		for (int i = 0; i < 3; i++)
			model[id + currentSample + i * size] = input[pixel_i + i];
	}

	randState[id] = localRandState;
}

__global__ void initBackgroundModelSamples(uint8_t* input, uint8_t* dest, uint8_t samplesCount) {
	constexpr int stride = 4;	// 4 bytes strides
	const int rgba_i = (blockDim.x * blockIdx.x + threadIdx.x) * stride;
	const int rgb_i = (blockDim.x * blockIdx.x + threadIdx.x) * (stride - 1) * samplesCount;

	for (int j = 0; j < samplesCount; j++) {
		dest[j * 3 + rgb_i] = input[rgba_i + 0];
		dest[j * 3 + rgb_i + 1] = input[rgba_i + 1];
		dest[j * 3 + rgb_i + 2] = input[rgba_i + 2];
	}
}

__global__ void initBackgroundModelSamples_new(uint8_t* input, uint8_t* dest, uint8_t samplesCount, int size) {
	constexpr int stride = 4;	// 4 bytes strides
	const int rgba_i = (blockDim.x * blockIdx.x + threadIdx.x) * stride;
	const int i = (blockDim.x * blockIdx.x + threadIdx.x);
	const int pixelffset = (i % size) * 3;

	for (int j = 0; j < samplesCount; j++) {
		for (int channel = 0; channel < 3; channel++)
			dest[j * size * 3 + i + channel * size] = input[rgba_i + channel];
	}
}

uint8_t* GPU::InsertionGraphicsPipeline::initialize(uint8_t * backgroundModel) {
	hipSetDevice(0);	// TODO try to move to constructor

	const int sampleSize = m_width * m_height * InsertionGraphicsPipeline::PIXEL_DATA_SIZE;
	// does not have alpha channel
	const int modelSampleSize = m_width * m_height * (InsertionGraphicsPipeline::PIXEL_DATA_SIZE - 1);
	constexpr uint8_t sampleCount = InsertionGraphicsPipeline::SAMPLE_COUNT;
	const int modelSize = modelSampleSize * sampleCount;

	uint8_t* d_input = nullptr;

	hipMalloc(reinterpret_cast<void**>(&m_d_model), modelSize);
	hipMalloc(reinterpret_cast<void**>(&d_input), sampleSize);
	hipMalloc(reinterpret_cast<void**>(&m_d_bgModel), modelSize);
	hipMemcpy(d_input, backgroundModel, sampleSize, hipMemcpyHostToDevice);

	initBackgroundModelSamples <<<900, 1024 >>> (d_input, m_d_model, sampleCount);
	initBackgroundModelSamples_new<<<900, 1024 >>> (d_input, m_d_bgModel, sampleCount, m_width * m_height);

	uint8_t* testOut = new uint8_t[m_width * m_height * 20 * 3];
	hipMemcpy(testOut, m_d_bgModel, m_width * m_height * 20 * 3, hipMemcpyDeviceToHost);
	uint8_t* sample = new uint8_t[m_width * m_height];
	int sample_id = 0;
	for (int i = 0; i < m_width * m_height; i++) {
		sample[i] = testOut[i];
	}

	return sample;
}

__global__ void randomy(RandState* states, float* out) {
	const int id = threadIdx.x;

	out[id] = devRand(states[id]);
}

void GPU::InsertionGraphicsPipeline::process(uint8_t * input, uint8_t * graphics, uint8_t * dest) {
	using T = decltype(input);
	T d_input = nullptr;
	T d_dest = nullptr;
	const uint32_t size = m_height * m_width;
	const size_t dataSize = size * 4;
	const size_t outputDataSize = size;
	RandState* d_randState = nullptr;
	RandState* randState = new uint32_t[size];
	
	srand(time(nullptr));
	for (int i = 0; i < size; i++)
		randState[i] = static_cast<uint32_t>(rand());

	hipSetDevice(0);
	hipMalloc(reinterpret_cast<void**>(&d_input), dataSize);
	hipMalloc(reinterpret_cast<void**>(&d_dest), outputDataSize);
	hipMalloc(reinterpret_cast<void**>(&d_randState), size * sizeof(RandState));

	hipMemcpy(d_input, input, dataSize, hipMemcpyHostToDevice);
	hipMemcpy(d_randState, randState, size * sizeof(uint32_t), hipMemcpyHostToDevice);

	segment_new<<<900, 1024>>> (d_input, m_d_model, d_dest);
	segment_fast<<<900, 1024 >>> (d_input, m_d_bgModel, d_dest, size, 0, d_randState);
	auto err = hipGetLastError();
	std::cout << hipGetErrorName(err);

	hipMemcpy(dest, d_dest, outputDataSize, hipMemcpyDeviceToHost);
	hipFree(d_input);
	hipFree(d_dest);
}