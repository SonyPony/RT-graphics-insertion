#include "hip/hip_runtime.h"
#include "gpuutil.cuh"
#include <stdlib.h>

void Gpu::Utils::generateRandStates(Gpu::Utils::RandState ** d_rs, size_t count)
{
    using namespace Gpu::Utils;

    auto h_randStates = new RandState[count];
    for (size_t i = 0; i < count; i++)
        h_randStates[i] = static_cast<uint32_t>(rand());

    // copy states to device
    hipMalloc(reinterpret_cast<void**>(d_rs), count * sizeof(RandState));
    hipMemcpy(*d_rs, h_randStates, count * sizeof(RandState), hipMemcpyHostToDevice);
}

__global__ void k_sobel_sep_v(uint8_t* input, short2* output) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x == 0 || x == FRAME_WIDTH - 1 || y == 0 || y == FRAME_HEIGHT - 1)
        return;

    const int kernelX[3] = {1, 2, 1};
    const int kernelY[3] = { -1, 0, 1 };
    short2 outputValue = {0, 0};

    for (int i = -1; i <= 1; i++) {
        const int inputValue = input[x + (y + i) * FRAME_WIDTH];

        outputValue.x += kernelX[i + 1] * inputValue;
        outputValue.y += kernelY[i + 1] * inputValue;
    }

    output[x + y * FRAME_WIDTH] = outputValue;
}

__global__ void k_sobel_sep_h(short2* input, short2* output) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x == 0 || x == FRAME_WIDTH - 1 || y == 0 || y == FRAME_HEIGHT - 1)
        return;

    const int kernelX[3] = { 1, 0, -1 };
    const int kernelY[3] = { 1, 2, 1 };
    short2 outputValue = { 0, 0 };

    for (int i = -1; i <= 1; i++) {
        const short2 inputValue = input[x + i + y * FRAME_WIDTH];

        outputValue.x += kernelX[i + 1] * inputValue.x;
        outputValue.y += kernelY[i + 1] * inputValue.y;
    }

    output[x + y * FRAME_WIDTH] = outputValue;
}

__global__ void k_testLab(uchar4* frame, uint8_t* out) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int id = x + y * FRAME_WIDTH;

    uchar4 pixel = frame[id];
    uint8_t lab = Gpu::Utils::cvtRGB2GRAY(pixel);

    out[id] = lab;
}

__global__ void k_dualcvtRGBA2RGB(uchar4* in1, uchar4* in2,
    uint8_t* out1, uint8_t* out2) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int id = x + y * FRAME_WIDTH;

    const uchar4 in1Pixel = in1[id];
    const uchar4 in2Pixel = in2[id];

    out1[id * 3] = in1Pixel.x;
    out1[id * 3 + 1] = in1Pixel.y;
    out1[id * 3 + 2] = in1Pixel.z;

    out2[id * 3] = in2Pixel.x;
    out2[id * 3 + 1] = in2Pixel.y;
    out2[id * 3 + 2] = in2Pixel.z;
}

__global__ void k_cvtRGBA2RGB_a(uchar4* in, uint8_t* out, uint8_t* outAlpha) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int id = x + y * FRAME_WIDTH;

    const uchar4 inPixel = in[id];

    out[id * 3] = inPixel.x;
    out[id * 3 + 1] = inPixel.y;
    out[id * 3 + 2] = inPixel.z;

    outAlpha[id] = inPixel.w;
}

__global__ void k_mirrorV(uchar4* in, uchar4* out) {
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int id = x + y * FRAME_WIDTH;

    out[x + (FRAME_HEIGHT - y - 1) * FRAME_WIDTH] = in[id];
}

void Gpu::Utils::gradients(dim3 dimGrid, dim3 dimBlock, uint8_t * input, short2 * temp, short2 * dest)
{
    k_sobel_sep_v << <dimGrid, dimBlock >> > (input, temp);
    k_sobel_sep_h << <dimGrid, dimBlock >> > (temp, dest);
}

void Gpu::Utils::dualCvtRGBA2RGB(dim3 dimGrid, dim3 dimBlock, uchar4* d_in1, uchar4* d_in2,
    uint8_t* d_out1, uint8_t* d_out2) {
    k_dualcvtRGBA2RGB << <dimGrid, dimBlock >> > (d_in1, d_in2, d_out1, d_out2);
}

void Gpu::Utils::cvtRGBA2RGB_A(dim3 dimGrid, dim3 dimBlock, uchar4 * d_in, uint8_t * d_out, uint8_t * d_outAlpha)
{
    k_cvtRGBA2RGB_a << <dimGrid, dimBlock >> > (d_in, d_out, d_outAlpha);
}

void Gpu::Utils::mirrorV(dim3 dimGrid, dim3 dimBlock, uchar4* d_in, uchar4* d_out) 
{
    k_mirrorV << <dimGrid, dimBlock >> > (d_in, d_out);
}
