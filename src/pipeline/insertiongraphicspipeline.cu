#include "hip/hip_runtime.h"
#include "insertiongraphicspipeline.cuh"
#include "../common/config.h"
#include "../pipeline/morphology/erosion.h"
#include "../pipeline/morphology/erosionFuncTemplate.h"
#include "../pipeline/segmentation/shadow_detector.cuh"
#include <QDebug>
#include "nppi_geometry_transforms.h"


void InsertionGraphicsPipeline::computeTransMatrix(cv::Size graphicsSize, cv::Point2f dstPoints[]) {
    m_graphicsSize = graphicsSize;
    cv::Point2f srcPoints[4];
    srcPoints[0] = cv::Point2f{ 0.f, 0.f };
    srcPoints[1] = cv::Point2f{ (float)graphicsSize.width, 0.f };
    srcPoints[3] = cv::Point2f{ 0.f, (float)graphicsSize.height };
    srcPoints[2] = cv::Point2f{ (float)graphicsSize.width, (float)graphicsSize.height };

    m_transformMat = cv::getPerspectiveTransform(srcPoints, dstPoints);
}

InsertionGraphicsPipeline::InsertionGraphicsPipeline() {
    hipMalloc(reinterpret_cast<void**>(&m_d_temp_C4_UC), FRAME_SIZE * 4);  // single channel

    m_graphicsSize = cv::Size{};

    m_transformMat = cv::Mat::eye(3, 3, CV_32F);
    m_segmenter = new ViBe(m_d_temp_C4_UC);
    m_shadowDectector = new ShadowDetector;
    m_trimapGenerator = new TrimapGenerator;
    m_matting = new GlobalSampling(m_d_temp_C4_UC);
    m_composer = new Composer(m_d_temp_C4_UC);
    m_bgHist = new BgHist;

    // alloc buffers on device
    hipMalloc(reinterpret_cast<void**>(&m_d_frame), FRAME_SIZE * Config::CHANNELS_COUNT_INPUT);
    hipMalloc(reinterpret_cast<void**>(&m_d_segmentation), FRAME_SIZE);    // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_trimap), FRAME_SIZE);  // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_shadowIntensity), FRAME_SIZE);  // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_graphicsAlphaMask), FRAME_SIZE);  // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_output), FRAME_SIZE * Config::CHANNELS_COUNT_INPUT);
    hipMalloc(reinterpret_cast<void**>(&m_d_transformedGraphics), FRAME_SIZE * Config::CHANNELS_COUNT_INPUT);

    m_d_rgbBg = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
    m_d_rgbFrame = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
    m_d_rgbGraphics = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
    m_d_labBg = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
    m_d_labFrame = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
    m_d_labGraphics = cv::cuda::createContinuous(FRAME_WIDTH, FRAME_HEIGHT, CV_8UC3);
}

InsertionGraphicsPipeline::~InsertionGraphicsPipeline()
{
    hipFree(m_d_frame);
    hipFree(m_d_segmentation);
    hipFree(m_d_temp_C4_UC);
    hipFree(m_d_trimap);
    hipFree(m_d_shadowIntensity);
    hipFree(m_d_graphicsAlphaMask);
    hipFree(m_d_output);
    hipFree(m_d_transformedGraphics);

    delete m_matting;
    delete m_shadowDectector;
    delete m_segmenter;
    delete m_trimapGenerator;
    delete m_composer;
    delete m_bgHist;
}

void InsertionGraphicsPipeline::initialize(Byte * frame)
{
    hipMemcpy(m_d_temp_C4_UC, frame, FRAME_SIZE * sizeof(uchar4), hipMemcpyHostToDevice);
    uchar4* d_bgInit = reinterpret_cast<uchar4*>(m_d_temp_C4_UC);
    m_segmenter->initialize(d_bgInit);
}

void InsertionGraphicsPipeline::process(Byte * input, Byte * graphics, Byte * output)
{
    hipSetDevice(0);
    dim3 dimGrid{ 80, 45 };
    dim3 dimBlock{ 16, 16 };

    // copy data
    hipMemcpy(m_d_frame, input, FRAME_SIZE * Config::CHANNELS_COUNT_INPUT, hipMemcpyHostToDevice);
    hipMemset(m_d_temp_C4_UC, 0, FRAME_SIZE * Config::CHANNELS_COUNT_INPUT);
    hipMemcpy(
        m_d_temp_C4_UC, 
        graphics, 
        GRAPHICS_WIDTH * GRAPHICS_HEIGHT * Config::CHANNELS_COUNT_INPUT, 
        hipMemcpyHostToDevice
    );
    
    // transform graphics
    cv::cuda::warpPerspective(
        cv::cuda::GpuMat{ m_graphicsSize, CV_8UC4, m_d_temp_C4_UC },
        cv::cuda::GpuMat{ cv::Size{ FRAME_WIDTH, FRAME_HEIGHT }, CV_8UC4, m_d_transformedGraphics },
        m_transformMat, cv::Size{ FRAME_WIDTH, FRAME_HEIGHT }, cv::INTER_NEAREST);
    
    uchar4* d_frame = reinterpret_cast<uchar4*>(m_d_frame);
    uchar4* d_graphics = reinterpret_cast<uchar4*>(m_d_transformedGraphics);

    // segmentation
    uchar4* d_background = m_segmenter->segment(d_frame, m_d_segmentation);

    // split alpha channel
    Gpu::Utils::dualCvtRGBA2RGB(
        dimGrid, dimBlock,
        d_frame, d_background,
        m_d_rgbFrame.ptr(), m_d_rgbBg.ptr());
    Gpu::Utils::cvtRGBA2RGB_A(
        dimGrid, dimBlock, d_graphics, m_d_rgbGraphics.ptr(), m_d_graphicsAlphaMask
    );

    // convert to LAB
    cv::cuda::cvtColor(m_d_rgbBg, m_d_labBg, cv::COLOR_RGB2Lab);
    cv::cuda::cvtColor(m_d_rgbFrame, m_d_labFrame, cv::COLOR_RGB2Lab);
    cv::cuda::cvtColor(m_d_rgbGraphics, m_d_labGraphics, cv::COLOR_RGB2Lab);
        
    // shadow segmentation
    m_shadowDectector->process(d_frame, m_d_segmentation, d_background,
        m_d_labFrame.ptr(), m_d_labBg.ptr(), m_d_shadowIntensity);

    // mophology refinement
    ErosionTemplateSharedTwoSteps(m_d_segmentation, m_d_temp_C4_UC, FRAME_WIDTH, FRAME_HEIGHT, 2);
    FilterDilation(m_d_segmentation, m_d_temp_C4_UC, FRAME_WIDTH, FRAME_HEIGHT, 2);

    FilterDilation(m_d_segmentation, m_d_temp_C4_UC, FRAME_WIDTH, FRAME_HEIGHT, 1);
    ErosionTemplateSharedTwoSteps(m_d_segmentation, m_d_temp_C4_UC, FRAME_WIDTH, FRAME_HEIGHT, 1);

    // trimap generation
    m_trimapGenerator->generate(m_d_segmentation, m_d_trimap);

    // image matting
    m_matting->matting(d_frame, m_d_trimap, d_background, m_d_segmentation);

    // assemble
    m_composer->compose(
        m_d_segmentation, m_d_shadowIntensity,
        m_d_rgbFrame.ptr(), m_d_labFrame.ptr(), m_d_labGraphics.ptr(), m_d_labBg.ptr(),
        m_d_graphicsAlphaMask, m_d_output
    );

    hipMemcpy(output, m_d_output, FRAME_SIZE * 3, hipMemcpyDeviceToHost);
}

void InsertionGraphicsPipeline::initAddFrame(Byte * frame)
{
    hipMemcpy(m_d_frame, frame, FRAME_SIZE * Config::CHANNELS_COUNT_INPUT, hipMemcpyHostToDevice);
    uchar4* d_frame = reinterpret_cast<uchar4*>(m_d_frame);

    m_bgHist->addFrame(d_frame);
}

void InsertionGraphicsPipeline::computeInitBg(uint8_t* output)
{
    uchar4* d_bgInit = reinterpret_cast<uchar4*>(m_d_temp_C4_UC);
    m_bgHist->computeMode(d_bgInit);
    m_segmenter->initialize(d_bgInit);
    m_bgHist->reset();
    hipMemcpy(output, m_d_temp_C4_UC, FRAME_SIZE * 4, hipMemcpyDeviceToHost);
}
