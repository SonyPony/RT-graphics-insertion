#include "insertiongraphicspipeline.cuh"
#include "../common/config.h"


Gpu::InsertionGraphicsPipeline::InsertionGraphicsPipeline(int width, int height) 
    : m_width{ width }, m_height{ height }, m_size{ height * width } {
    m_segmenter = new ViBe(width, height);

    // alloc buffers on device
    hipMalloc(reinterpret_cast<void**>(&m_d_frame), m_size * Config::CHANNELS_COUNT_INPUT);
    hipMalloc(reinterpret_cast<void**>(&m_d_segmentation), m_size);    // single channel

}

Gpu::InsertionGraphicsPipeline::~InsertionGraphicsPipeline()
{
    hipFree(m_d_frame);
    hipFree(m_d_segmentation);
    delete m_segmenter;
}

void Gpu::InsertionGraphicsPipeline::initialize(Byte * frame)
{
    m_segmenter->initialize(frame);
}

void Gpu::InsertionGraphicsPipeline::process(Byte * input, Byte * graphics, Byte * output)
{
    hipSetDevice(0);

    // segmentation
    hipMemcpy(m_d_frame, input, m_size * Config::CHANNELS_COUNT_INPUT, hipMemcpyHostToDevice);
    m_segmenter->segment(m_d_frame, m_d_segmentation);
    hipMemcpy(output, m_d_segmentation, m_size, hipMemcpyDeviceToHost);

    // matting
}
