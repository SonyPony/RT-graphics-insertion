#include "hip/hip_runtime.h"
#include "insertiongraphicspipeline.cuh"
#include "../common/config.h"
#include "../pipeline/morphology/erosion.h"
#include "../pipeline/morphology/erosionFuncTemplate.h"
#include "../pipeline/trimap/trimap_generator.cuh"
#include "../pipeline/segmentation/shadow_detector.cuh"

#include <npp.h>

Gpu::InsertionGraphicsPipeline::InsertionGraphicsPipeline() {
    hipMalloc(reinterpret_cast<void**>(&m_d_temp_C4_UC), FRAME_SIZE * 4);  // single channel

    m_segmenter = new ViBe(m_d_temp_C4_UC);
    m_matting = new GlobalSampling;

    // alloc buffers on device
    hipMalloc(reinterpret_cast<void**>(&m_d_frame), FRAME_SIZE * Config::CHANNELS_COUNT_INPUT);
    hipMalloc(reinterpret_cast<void**>(&m_d_segmentation), FRAME_SIZE);    // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_trimap), FRAME_SIZE);  // single channel
    

}

Gpu::InsertionGraphicsPipeline::~InsertionGraphicsPipeline()
{
    hipFree(m_d_frame);
    hipFree(m_d_segmentation);
    hipFree(m_d_temp_C4_UC);
    hipFree(m_d_trimap);

    delete m_matting;
    delete m_segmenter;
}

void Gpu::InsertionGraphicsPipeline::initialize(Byte * frame)
{
    m_segmenter->initialize(frame);
}

void Gpu::InsertionGraphicsPipeline::process(Byte * input, Byte * graphics, Byte * output)
{
    hipSetDevice(0);

    uchar4* d_frame = reinterpret_cast<uchar4*>(m_d_frame);

    // segmentation
    hipMemcpy(m_d_frame, input, FRAME_SIZE * Config::CHANNELS_COUNT_INPUT, hipMemcpyHostToDevice);
    /*Byte* d_background = */m_segmenter->segment(d_frame, m_d_segmentation);
    hipMemcpy(output, m_d_segmentation, FRAME_SIZE, hipMemcpyDeviceToHost);


    uint8_t* d_shadowIntensity;
    hipMalloc(reinterpret_cast<void**>(&d_shadowIntensity), FRAME_SIZE);
    uint8_t* d_trimap;
    hipMalloc(reinterpret_cast<void**>(&d_trimap), FRAME_SIZE);

    uint8_t* d_temp;
    uint8_t* d_temp2;
    hipMalloc(reinterpret_cast<void**>(&d_temp), FRAME_SIZE);
    hipMalloc(reinterpret_cast<void**>(&d_temp2), FRAME_SIZE);
    hipMemcpy(d_temp2, m_d_segmentation, FRAME_SIZE, hipMemcpyDeviceToDevice);

    
    
    
    // trimap
    /*TrimapGenerator trimapGen;
    hipMemcpy(d_temp, m_d_segmentation, m_size, hipMemcpyDeviceToDevice);
    trimapGen.generate(d_temp, m_d_segmentation);*/

    // matting
    //hipMemcpy(m_d_trimap, trimap, FRAME_SIZE, hipMemcpyHostToDevice);
    //m_matting->matting(m_d_frame, m_d_trimap, d_background, m_d_segmentation);

    dim3 dimGrid{ 80, 45 };
    dim3 dimBlock{ 16, 16 };

    uchar4* d_frame = reinterpret_cast<uchar4*>(m_d_frame);
    //hipMemcpy(m_d_segmentation, d_temp2, m_size, hipMemcpyDeviceToDevice);
    ShadowDetector shadowDectector;
    shadowDectector.process(d_frame, m_d_segmentation, d_background, d_shadowIntensity);

    
    ErosionTemplateSharedTwoSteps(m_d_segmentation, d_temp, FRAME_WIDTH, FRAME_HEIGHT, 2);
    FilterDilation(m_d_segmentation, d_temp, FRAME_WIDTH, FRAME_HEIGHT, 2);


    FilterDilation(m_d_segmentation, d_temp, FRAME_WIDTH, FRAME_HEIGHT, 1);
    ErosionTemplateSharedTwoSteps(m_d_segmentation, d_temp, FRAME_WIDTH, FRAME_HEIGHT, 1);

    TrimapGenerator trimapGen;
    trimapGen.generate(m_d_segmentation, d_trimap);

    m_matting->matting(m_d_frame, d_trimap, d_background, m_d_segmentation);

    // TEST output
    hipMemcpy(output, m_d_segmentation, FRAME_SIZE, hipMemcpyDeviceToHost);
}
