#include "insertiongraphicspipeline.cuh"
#include "../common/config.h"


Gpu::InsertionGraphicsPipeline::InsertionGraphicsPipeline(int width, int height) 
    : m_width{ width }, m_height{ height }, m_size{ height * width } {
    m_segmenter = new ViBe(width, height);
    m_matting = new GlobalSampling(width, height);

    // alloc buffers on device
    hipMalloc(reinterpret_cast<void**>(&m_d_frame), m_size * Config::CHANNELS_COUNT_INPUT);
    hipMalloc(reinterpret_cast<void**>(&m_d_segmentation), m_size);    // single channel
    hipMalloc(reinterpret_cast<void**>(&m_d_trimap), m_size);  // single channel

}

Gpu::InsertionGraphicsPipeline::~InsertionGraphicsPipeline()
{
    hipFree(m_d_frame);
    hipFree(m_d_segmentation);
    delete m_segmenter;
}

void Gpu::InsertionGraphicsPipeline::initialize(Byte * frame)
{
    m_segmenter->initialize(frame);
}

void Gpu::InsertionGraphicsPipeline::process(Byte * input, Byte * graphics, Byte * output, /*temp*/ Byte* trimap)
{
    hipSetDevice(0);

    // segmentation
    hipMemcpy(m_d_frame, input, m_size * Config::CHANNELS_COUNT_INPUT, hipMemcpyHostToDevice);
    m_segmenter->segment(m_d_frame, m_d_segmentation);
    hipMemcpy(output, m_d_segmentation, m_size, hipMemcpyDeviceToHost);

    // matting
    hipMemcpy(m_d_trimap, trimap, m_size, hipMemcpyHostToDevice);
    m_matting->matting(m_d_frame, m_d_trimap, m_d_segmentation);

    // TEST output
    hipMemcpy(output, m_d_segmentation, m_size, hipMemcpyDeviceToHost);
}
